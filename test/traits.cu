#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>

int main(void)
{
	// generate 32M random numbers serially
	thrust::host_vector<int> h_vec(10000);
	std::generate(h_vec.begin(), h_vec.end(), rand);

	//std::sort(h_vec.begin(), h_vec.end());

	// transfer data to the device
	thrust::device_vector<int> d_vec = h_vec;

	// sort data on the device (846M keys per second on GeForce GTX 480)
	thrust::sort(d_vec.begin(), d_vec.end());

	// transfer data back to host
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	std::cout << h_vec.size() << ", " << std::is_sorted(h_vec.begin(), h_vec.end());

	return 0;
}